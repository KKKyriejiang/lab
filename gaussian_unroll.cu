#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>

// 错误检查宏
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

// 定义循环展开因子
#define UNROLL_FACTOR 4

/**
 * @brief CUDA 核函数：执行行除法
 */
__global__ void division_kernel(float *d_m, int n, int k) {
    int j = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    float pivot_val = d_m[k * (n + 1) + k];
    if (j <= n) {
        d_m[k * (n + 1) + j] /= pivot_val;
    }
}

/**
 * @brief CUDA 核函数：使用共享内存瓦片和循环展开执行消元
 * @tparam BLOCK_DIM 线程块的维度
 */
template <int BLOCK_DIM>
__global__ void elimination_kernel_unrolled(float *d_m, int n, int k) {
    // 定义共享内存瓦片
    __shared__ float factor_tile[BLOCK_DIM];

    // 计算全局行索引
    int row = k + 1 + blockIdx.y * BLOCK_DIM + threadIdx.y;

    // 1. 加载消元因子到共享内存
    if (row < n && threadIdx.x == 0) { // 每行的第一个线程加载因子
        factor_tile[threadIdx.y] = d_m[row * (n + 1) + k];
    }
    __syncthreads();

    // 2. 循环展开计算
    if (row < n) {
        float factor = factor_tile[threadIdx.y];
        int col_start = k + 1 + blockIdx.x * BLOCK_DIM * UNROLL_FACTOR + threadIdx.x;

        // 使用 #pragma unroll 指示编译器展开此循环
        #pragma unroll
        for (int i = 0; i < UNROLL_FACTOR; ++i) {
            int col = col_start + i * BLOCK_DIM;
            if (col <= n) {
                float pivot_val = d_m[k * (n + 1) + col]; // 主元行仍然从全局内存读取
                d_m[row * (n + 1) + col] -= factor * pivot_val;
            }
        }
    }
    
    __syncthreads();

    // 3. 内核融合：置零
    if (row < n && threadIdx.x == 0) {
        d_m[row * (n + 1) + k] = 0.0f;
    }
}


// 在 CPU 上初始化矩阵和向量
void init_matrix(std::vector<float>& m, int n) {
    srand(time(0));
    for (int i = 0; i < n; ++i) {
        float row_sum = 0.0f;
        for (int j = 0; j < n; ++j) {
            m[i * (n + 1) + j] = static_cast<float>(rand()) / RAND_MAX * 10.0f;
            row_sum += fabs(m[i * (n + 1) + j]);
        }
        m[i * (n + 1) + i] += row_sum; 
        m[i * (n + 1) + n] = static_cast<float>(rand()) / RAND_MAX * 10.0f;
    }
}

// 在 CPU 上执行高斯消去（用于基准测试）
void cpu_gaussian_elimination(std::vector<float>& m, int n) {
    for (int k = 0; k < n; ++k) {
        float pivot = m[k * (n + 1) + k];
        for (int j = k; j <= n; ++j) {
            m[k * (n + 1) + j] /= pivot;
        }
        for (int i = k + 1; i < n; ++i) {
            float factor = m[i * (n + 1) + k];
            for (int j = k; j <= n; ++j) {
                m[i * (n + 1) + j] -= factor * m[k * (n + 1) + j];
            }
        }
    }
}

// 启动器函数
void launch_elimination_kernel(float* d_m, int n, int k, int block_dim) {
    dim3 block(block_dim, block_dim);
    // 因为有循环展开，x维度的网格可以相应减小
    dim3 grid(
        ( (n + 1 - (k + 1)) + (block.x * UNROLL_FACTOR) - 1) / (block.x * UNROLL_FACTOR),
        (n - (k + 1) + block.y - 1) / block.y
    );

    switch (block_dim) {
        case 8:
            elimination_kernel_unrolled<8><<<grid, block>>>(d_m, n, k);
            break;
        case 16:
            elimination_kernel_unrolled<16><<<grid, block>>>(d_m, n, k);
            break;
        case 32:
            elimination_kernel_unrolled<32><<<grid, block>>>(d_m, n, k);
            break;
        default:
            std::cerr << "Unsupported block dimension: " << block_dim << std::endl;
            exit(1);
    }
}


int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size_N> <block_dim>" << std::endl;
        std::cerr << "Supported block_dim: 8, 16, 32" << std::endl;
        return 1;
    }

    const int N = std::atoi(argv[1]);
    const int block_dim = std::atoi(argv[2]);

    if (N <= 0 || (block_dim != 8 && block_dim != 16 && block_dim != 32)) {
        std::cerr << "Invalid arguments." << std::endl;
        return 1;
    }

    const int matrix_rows = N;
    const int augmented_cols = N + 1;
    const size_t matrix_size_bytes = matrix_rows * augmented_cols * sizeof(float);

    // --- CPU 版本 ---
    std::vector<float> h_m_cpu(matrix_rows * augmented_cols);
    init_matrix(h_m_cpu, matrix_rows);
    
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gaussian_elimination(h_m_cpu, matrix_rows);
    auto cpu_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_stop - cpu_start;
    std::cout << "CPU_TIME:" << cpu_duration.count() << std::endl;

    // --- GPU 版本 ---
    std::vector<float> h_m_gpu(matrix_rows * augmented_cols);
    init_matrix(h_m_gpu, matrix_rows);

    float* d_m = nullptr;
    CUDA_CHECK(hipMalloc(&d_m, matrix_size_bytes));
    CUDA_CHECK(hipMemcpy(d_m, h_m_gpu.data(), matrix_size_bytes, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));

    for (int k = 0; k < matrix_rows; ++k) {
        // --- 除法步骤 ---
        int div_threads_needed = augmented_cols - 1 - k;
        int div_grid_size = (div_threads_needed + block_dim - 1) / block_dim;
        division_kernel<<<div_grid_size, block_dim>>>(d_m, matrix_rows, k);
        CUDA_CHECK(hipGetLastError());
        
        float one = 1.0f;
        CUDA_CHECK(hipMemcpy(d_m + k * augmented_cols + k, &one, sizeof(float), hipMemcpyHostToDevice));

        // --- 消元步骤 (使用共享内存和循环展开) ---
        if (matrix_rows - 1 - k > 0) {
            launch_elimination_kernel(d_m, matrix_rows, k, block_dim);
            CUDA_CHECK(hipGetLastError());
        }
    }
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "GPU_TIME:" << milliseconds << std::endl;

    CUDA_CHECK(hipFree(d_m));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}