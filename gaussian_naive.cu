#include <iostream>
#include <vector>
#include <cstdlib>
#include <ctime>
#include <cmath>
#include <iomanip>
#include <chrono>
#include <hip/hip_runtime.h>

// CUDA 块大小
#define BLOCK_SIZE 256

// 错误检查宏
#define CUDA_CHECK(err) { \
    hipError_t err_ = (err); \
    if (err_ != hipSuccess) { \
        std::cerr << "CUDA Error at " << __FILE__ << ":" << __LINE__ \
                  << " - " << hipGetErrorString(err_) << std::endl; \
        exit(EXIT_FAILURE); \
    } \
}

/**
 * @brief CUDA 核函数：执行行除法 (保持不变)
 */
__global__ void division_kernel(float *d_m, int n, int k) {
    int j = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if (j <= n) {
        float pivot_val = d_m[k * (n + 1) + k];
        d_m[k * (n + 1) + j] /= pivot_val;
    }
}

/**
 * @brief CUDA 核函数：执行消元步骤 (平凡实现)
 * 每个线程负责计算一个元素的更新。
 * 所有数据均从全局内存直接读取，无共享内存优化。
 */
__global__ void elimination_kernel_naive(float *d_m, int n, int k) {
    // 二维线程索引映射到矩阵的行和列
    int i = k + 1 + blockIdx.y * blockDim.y + threadIdx.y;
    int j = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n && j <= n) {
        // 每个线程都从全局内存读取消元因子和主元行值
        // 这会导致对同一数据的大量重复读取
        float factor = d_m[i * (n + 1) + k];
        float pivot_row_val = d_m[k * (n + 1) + j];
        d_m[i * (n + 1) + j] -= factor * pivot_row_val;
    }
}

/**
 * @brief CUDA 核函数：将消元后的列元素置零
 */
__global__ void set_zeros_kernel(float* d_m, int n, int k) {
    int i = k + 1 + blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        d_m[i * (n + 1) + k] = 0.0f;
    }
}

// 在 CPU 上初始化矩阵和向量
void init_matrix(std::vector<float>& m, int n) {
    srand(time(0));
    for (int i = 0; i < n; ++i) {
        float row_sum = 0.0f;
        for (int j = 0; j < n; ++j) {
            m[i * (n + 1) + j] = static_cast<float>(rand()) / RAND_MAX * 10.0f;
            row_sum += fabs(m[i * (n + 1) + j]);
        }
        m[i * (n + 1) + i] += row_sum; 
        m[i * (n + 1) + n] = static_cast<float>(rand()) / RAND_MAX * 10.0f;
    }
}

// 在 CPU 上执行高斯消去（用于基准测试）
void cpu_gaussian_elimination(std::vector<float>& m, int n) {
    for (int k = 0; k < n; ++k) {
        float pivot = m[k * (n + 1) + k];
        for (int j = k; j <= n; ++j) {
            m[k * (n + 1) + j] /= pivot;
        }
        for (int i = k + 1; i < n; ++i) {
            float factor = m[i * (n + 1) + k];
            for (int j = k; j <= n; ++j) {
                m[i * (n + 1) + j] -= factor * m[k * (n + 1) + j];
            }
        }
    }
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << "Usage: " << argv[0] << " <matrix_size_N>" << std::endl;
        return 1;
    }

    const int N = std::atoi(argv[1]);
    if (N <= 0) {
        std::cerr << "Matrix size must be a positive integer." << std::endl;
        return 1;
    }

    const int matrix_rows = N;
    const int augmented_cols = N + 1;
    const size_t matrix_size_bytes = matrix_rows * augmented_cols * sizeof(float);

    // --- CPU 版本 ---
    std::vector<float> h_m_cpu(matrix_rows * augmented_cols);
    init_matrix(h_m_cpu, matrix_rows);
    
    auto cpu_start = std::chrono::high_resolution_clock::now();
    cpu_gaussian_elimination(h_m_cpu, matrix_rows);
    auto cpu_stop = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> cpu_duration = cpu_stop - cpu_start;
    std::cout << "CPU_TIME:" << cpu_duration.count() << std::endl;

    // --- GPU 版本 ---
    std::vector<float> h_m_gpu(matrix_rows * augmented_cols);
    init_matrix(h_m_gpu, matrix_rows);

    float* d_m = nullptr;
    CUDA_CHECK(hipMalloc(&d_m, matrix_size_bytes));
    CUDA_CHECK(hipMemcpy(d_m, h_m_gpu.data(), matrix_size_bytes, hipMemcpyHostToDevice));

    hipEvent_t start, stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));
    
    CUDA_CHECK(hipEventRecord(start));

    for (int k = 0; k < matrix_rows; ++k) {
        // --- 除法步骤 ---
        int div_threads_needed = augmented_cols - 1 - k;
        int div_grid_size = (div_threads_needed + BLOCK_SIZE - 1) / BLOCK_SIZE;
        division_kernel<<<div_grid_size, BLOCK_SIZE>>>(d_m, matrix_rows, k);
        CUDA_CHECK(hipGetLastError());
        
        float one = 1.0f;
        CUDA_CHECK(hipMemcpy(d_m + k * augmented_cols + k, &one, sizeof(float), hipMemcpyHostToDevice));

        // --- 消元步骤 (平凡实现) ---
        if (matrix_rows - 1 - k > 0) {
            dim3 block(16, 16); // 使用 16x16 的块
            dim3 grid(
                (augmented_cols - 1 - k + block.x - 1) / block.x,
                (matrix_rows - 1 - k + block.y - 1) / block.y
            );
            elimination_kernel_naive<<<grid, block>>>(d_m, matrix_rows, k);
            CUDA_CHECK(hipGetLastError());

            // --- 置零步骤 (单独调用) ---
            int zero_rows_needed = matrix_rows - 1 - k;
            int zero_grid_size = (zero_rows_needed + BLOCK_SIZE - 1) / BLOCK_SIZE;
            set_zeros_kernel<<<zero_grid_size, BLOCK_SIZE>>>(d_m, matrix_rows, k);
            CUDA_CHECK(hipGetLastError());
        }
    }
    
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));

    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    std::cout << "GPU_TIME:" << milliseconds << std::endl;

    CUDA_CHECK(hipFree(d_m));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));

    return 0;
}